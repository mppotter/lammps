#include "hip/hip_runtime.h"
// **************************************************************************
//                           lj_bump_smooth_linear.cu
//                             -------------------
//                              Mark Potter (RPI)
//
//  Device code for acceleration of the lj/bump/smooth/linear pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : pottem3@rpi.edu
// ***************************************************************************


#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#include <math.h>
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
#else
_texture_2d( pos_tex,int4);
#endif
#else
#define pos_tex x_
#endif

__kernel void k_lj_bump_smooth_linear(const __global numtyp4 *restrict x_,
                        const __global numtyp4 *restrict lj1,
                        const __global numtyp4 *restrict lj3,
                        const int lj_types,
                        const __global numtyp *restrict sp_lj,
                        const __global numtyp4 *restrict bump_data,
                        const __global numtyp4 *restrict smooth_linear_data,
                        const __global int * dev_nbor,
                        const __global int * dev_packed,
                        __global acctyp4 *restrict ans,
                        __global acctyp *restrict engv,
                        const int eflag, const int vflag, const int inum,
                        const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_pair();

  // for bump
  numtyp rtmp, btmp;
  numtyp rinv;

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (r2inv<lj1[mtype].z) {
        r2inv = ucl_recip(r2inv); // This makes r2inv the actual r^2 inverse
        rtmp = rsqrt(r2inv);
        rinv = ucl_recip(rtmp);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        force = rinv*force - smooth_linear_data[mtype].y;
        force*=factor_lj*rinv;

        // for bump
        if(rtmp >= bump_data[mtype].x && rtmp <= bump_data[mtype].y) {
            force += -bump_data[mtype].z*M_PI*sinpi((bump_data[mtype].y+bump_data[mtype].x-rtmp-rtmp)/(bump_data[mtype].y-bump_data[mtype].x))/(bump_data[mtype].y-bump_data[mtype].x)/rtmp;
        }

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
          numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          e-=smooth_linear_data[mtype].x
             +(rtmp-smooth_linear_data[mtype].z)*smooth_linear_data[mtype].y;
          //bump
          if(rtmp >= bump_data[mtype].x && rtmp <= bump_data[mtype].y) {
            btmp = sinpi((bump_data[mtype].y-rtmp)/(bump_data[mtype].y-bump_data[mtype].x));
            e += bump_data[mtype].z*btmp*btmp;
          }
          // energy+=factor_lj*(e-lj3[mtype].z);
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

__kernel void k_lj_bump_smooth_linear_fast(const __global numtyp4 *restrict x_,
                        const __global numtyp4 *restrict lj1_in,
                        const __global numtyp4 *restrict lj3_in,
                        const __global numtyp *restrict sp_lj_in,
                        const __global numtyp4 *restrict bump_data_in,
                        const __global numtyp4 *restrict smooth_linear_data_in,
                        const __global int * dev_nbor,
                        const __global int * dev_packed,
                        __global acctyp4 *restrict ans,
                        __global acctyp *restrict engv,
                        const int eflag, const int vflag, const int inum,
                        const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  // for bump
  numtyp rtmp, btmp;
  numtyp rinv;
  #ifndef ONETYPE
  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 bump_data[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 smooth_linear_data[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    bump_data[tid]=bump_data_in[tid];
    smooth_linear_data[tid] = smooth_linear_data_in[tid];
    if (EVFLAG && eflag)
      lj3[tid]=lj3_in[tid];
  }
  __syncthreads();
  #else
  const numtyp lj1x=lj1_in[ONETYPE].x;
  const numtyp lj1y=lj1_in[ONETYPE].y;
  const numtyp cutsq=lj1_in[ONETYPE].z;
  const numtyp bumpx=bump_data_in[ONETYPE].x;
  const numtyp bumpy=bump_data_in[ONETYPE].y;
  const numtyp bumpz=bump_data_in[ONETYPE].z;
  const numtyp slx=smooth_linear_data_in[ONETYPE].x;
  const numtyp sly=smooth_linear_data_in[ONETYPE].y;
  const numtyp slz=smooth_linear_data_in[ONETYPE].z;
  numtyp lj3x, lj3y, lj3z;
  if (EVFLAG && eflag) {
    lj3x=lj3_in[ONETYPE].x;
    lj3y=lj3_in[ONETYPE].y;
    lj3z=lj3_in[ONETYPE].z;
  }
  #endif

  int n_stride;
  local_allocate_store_pair();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    #ifndef ONETYPE
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    numtyp factor_lj;
    #endif

    NOUNROLL
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      #ifndef ONETYPE
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      numtyp cutsq=lj1[mtype].z;
      #endif

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      if (r2inv<cutsq) {
        #ifndef ONETYPE
        numtyp lj1x=lj1[mtype].x;
        numtyp lj1y=lj1[mtype].y;
        numtyp bumpx=bump_data[mtype].x;
        numtyp bumpy=bump_data[mtype].y;
        numtyp bumpz=bump_data[mtype].z;
        numtyp slx=smooth_linear_data[mtype].x;
        numtyp sly=smooth_linear_data[mtype].y;
        numtyp slz=smooth_linear_data[mtype].z;
        #endif
        r2inv = ucl_recip(r2inv); // This makes r2inv the actual r^2 inverse
        rtmp = rsqrt(r2inv);
        rinv = ucl_recip(rtmp);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r6inv*(lj1x*r6inv-lj1y);
        force = rinv*force - sly;
        #ifndef ONETYPE
        force*=factor_lj*rinv;
        #endif

        // for bump
        if(rtmp >= bumpx && rtmp <= bumpy) {
            force += -bumpz*M_PI*sinpi((bumpy+bumpx-rtmp-rtmp)/(bumpy-bumpx))/(bumpy-bumpx)/rtmp;
        }

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (EVFLAG && eflag) {
            #ifndef ONETYPE
            numtyp lj3x=lj3[mtype].x;
            numtyp lj3y=lj3[mtype].y;
            numtyp lj3z=lj3[mtype].z;
            #endif
            numtyp e=r6inv*(lj3x*r6inv-lj3y);
            e-=slx+(rtmp-slz)*sly;
            //bump
            if(rtmp >= bumpx && rtmp <= bumpy) {
                btmp = sinpi((bumpy-rtmp)/(bumpy-bumpx));
                e += bumpz*btmp*btmp;
            }
          #ifndef ONETYPE
          energy+=factor_lj*(e-lj3z);
          #else
          energy+=(e-lj3z);
          #endif
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

